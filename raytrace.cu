#include "hip/hip_runtime.h"
#include "raytrace.cuh"

/*------------Template Declarations---------------*/


__device__ Sphere *cuda_scene;
__device__ Light *cuda_lights;

#define HEIGHT 1024
#define WIDTH 1024



void ExitCleanup()
{
	//hipFree(cuda_scene);
	//hipFree(lights);
}


__device__ bool IntersectionBounds(float T, float t_min, float t_max) {
	return (T > t_min && T < t_max);  // Strict inequality
}

__device__ float4 ReflectRay(const float4& R, const float4& N)
{
	return ((N * (N.dot(R))) * 2) - R;
}

__device__ double CalcLight(const float4& P, const float4& N, const float4& V, const int& s)
{
	double intensity = 0.0;
	double t_max = 0;
	float4 L = {};
	float4 R = {};
	for (int i = 0; i < sizeof(cuda_lights) / sizeof(Light); i++)
	{
		if (cuda_lights[i].type == cuda_lights->AMBIENT)
		{
			intensity += cuda_lights[i].intensity;
		}
		else
		{
			if (cuda_lights[i].type == cuda_lights->POINT)
			{
				L = (cuda_lights[i].pos - P);
				t_max = 1;
			}
			else
			{
				L = cuda_lights[i].pos;
				t_max = INFINITY;
			}
			L = L;
			Intersection result_closest = ClosestIntersection(P, L, 0.00001, t_max);
			if (result_closest.closest_sphere != NULL)
			{
				continue;
			}

			double n_dot_l = N.dot(L);
			if (n_dot_l > 0)
			{
				intensity += cuda_lights[i].intensity * n_dot_l / (N.len() * L.len());
			}

			if (s != -1)
			{
				R = ReflectRay(L, N);
				double r_dot_v = R.dot(V);

				if (r_dot_v > 0)
				{
					intensity += cuda_lights[i].intensity * pow(r_dot_v / (R.len() * (V.len())), s);
				}

			}
		}
	}
	return intensity;
}

__device__ double IntersectRaySphere(const float4& O, const float4& D, const Sphere& sphere, const double& dDot)
{
	float4 CO = {};
	CO = O - sphere.center;
	
	double a = dDot;
	double b = 2 * CO.dot(D);
	double c = CO.dot(CO) - sphere.sRadius;

	double discr = b * b - 4 * a * c;

	if (discr < 0)
	{
		return INFINITY;
	}
	else if (discr == 0)
	{
		return -b / (2 * a);
	}

	double t = (-b - sqrt(discr)) / (2 * a);		// Minimize compute only go for 1 root;

	return t;
}

__device__ Intersection ClosestIntersection(const float4& O, const float4& D, const double& t_min, const double& t_max)
{
	double closest_t = INFINITY;
	Sphere* closest_sphere = NULL;
	double d_dot_d = D.dot(D);		// Cache immutable value


	for (int i = 0; i < sizeof(cuda_scene) / sizeof(Sphere); i++)
	{
		double t = IntersectRaySphere(O, D, cuda_scene[i], d_dot_d);

		if (IntersectionBounds(t, t_min, t_max) && t < closest_t)
		{
			closest_t = t;
			closest_sphere = const_cast<Sphere*>(&cuda_scene[i]);
		}
	}
	return Intersection(closest_sphere, closest_t);
}

__device__ COLORREF TraceRay(const float4& O, const float4& D, const double& t_min, const double& t_max, const int& recursionDepth)
{
	float4 N = {};
	float4 P = {};
	float4 R = {};

	Intersection result_closest = ClosestIntersection(O,D,t_min,t_max);
	Sphere *closest_sphere = result_closest.closest_sphere;
	double closest_t = result_closest.closest_t;

	if (closest_sphere == NULL)
	{
		return RGB(0, 0, 0);
	}

	P = O + (D * closest_t);
	N = (P - closest_sphere->center).norm();

	double res = CalcLight(P, N, D.invert(), closest_sphere->specularity);
	int r = (int)round(GetRValue(closest_sphere->color) * res);
	int g = (int)round(GetGValue(closest_sphere->color) * res);
	int b = (int)round(GetBValue(closest_sphere->color) * res);

	double refl = closest_sphere->reflective;

	if (recursionDepth <= 0 || refl <= 0)
	{
		return RGB(max(0, min(255, r)),
			max(0, min(255, g)),
			max(0, min(255, b)));
	}


	R = ReflectRay(D.invert(), N);
	COLORREF reflectedColor = TraceRay(P, R, t_min, t_max, recursionDepth - 1);

	int reflected_r = (int)round(GetRValue(reflectedColor)) * refl;
	int reflected_g = (int)round(GetGValue(reflectedColor)) * refl;
	int reflected_b = (int)round(GetBValue(reflectedColor)) * refl;


	return RGB(max(0, min(255, static_cast<int>(r * (1 - refl) + reflected_r))),
		max(0, min(255, static_cast<int>(g * (1 - refl) + reflected_g))),
		max(0, min(255, static_cast<int>(b * (1 - refl) + reflected_b))));

}

__device__ float4 CanvasToViewPort(const int &x, const int &y)
{
	// for simplicity : Vw = Vh = d = 1    approx 53 fov
	double aspectRatio = static_cast<double>(WIDTH) / HEIGHT;

	// Map x and y to the viewport, adjusting by aspect ratio
	double fovMod = 1;
	double viewportX = (x - WIDTH / 2.0) * ((1.0 * fovMod) / WIDTH) * aspectRatio;
	double viewportY = -(y - HEIGHT/ 2.0) * ((1.0 * fovMod) / HEIGHT); // Flip Y to match 3D space orientation

	return float4(viewportX, viewportY, 1);  // Z=1 for perspective projection
}

__global__ void cuda_Draw(BYTE *pLpvBits, Camera &cam)
{
	float4 D = {};
	float4 N = {};
	float4 P = {};
	double t_min = 0.0001;
	double t_max = INFINITY;
	int recursionDepth = 2;

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;


	D = CanvasToViewPort(x, y).norm();
	D = cam.ApplyCameraRotation(D, cam).norm();
	COLORREF color = TraceRay(cam.position, D, t_min, t_max, recursionDepth);
	D = normalize(D);


	int offset = (y * WIDTH + x) * 4;
	if (offset >= 0 && offset < WIDTH * HEIGHT * 4 - 4)
	{
		pLpvBits[offset + 0] = (int)GetBValue(color);
		pLpvBits[offset + 1] = (int)GetGValue(color);
		pLpvBits[offset + 2] = (int)GetRValue(color);
		pLpvBits[offset + 3] = 255;
	}

}


__host__ void Draw_Caller(BYTE ** pLpvBits, Camera &cam, Sphere *&scene, Light *&lights)
{
	int buffer_size = WIDTH * HEIGHT * sizeof(BYTE) * 4;

	int N = 1024;

	dim3 threadsPB(16,16);
	dim3 numB(N/threadsPB.x, N / threadsPB.y);

	BYTE *cuda_lpvbits;
	size_t src_pitch = ((WIDTH * 4 + 3) & ~3);	// AND with (NOT 3) ensures last two digits are always 0
	size_t dest_pitch;

	hipMallocPitch(&cuda_lpvbits,
					&dest_pitch,
					WIDTH * 4 * sizeof(BYTE),	// 4 bytes for each pixel; R, G, B, alpha
					HEIGHT);					// number of rows

	hipMemcpy2D(cuda_lpvbits,					// Destinaion
				 dest_pitch,
				 *pLpvBits,						// Source
				 src_pitch,
				 WIDTH * 4 * sizeof(BYTE),
				 HEIGHT,
				 hipMemcpyHostToDevice);



	Camera *cuda_cam;

	hipMalloc(&cuda_cam, sizeof(Camera));
	hipMemcpy(cuda_cam, 
			   &cam, 
			   sizeof(Camera), 
			   hipMemcpyHostToDevice);
	


	int number_of_spheres = 3;
	int number_of_lights = 2;

	hipMalloc(&cuda_scene, (number_of_spheres * sizeof(Sphere)));
	hipMalloc(&cuda_lights, (number_of_lights * sizeof(Light)));

	hipMemcpyToSymbol(HIP_SYMBOL(cuda_scene),
					   scene,
					   number_of_spheres * sizeof(Sphere),
					   0,
					   hipMemcpyHostToDevice);
	
	hipMemcpyToSymbol(HIP_SYMBOL(cuda_lights),
					   lights,
					   number_of_lights * sizeof(Light),
					   0,
					   hipMemcpyHostToDevice);

	
	cuda_Draw<<<numB, threadsPB>>>(cuda_lpvbits, *cuda_cam);

	hipDeviceSynchronize();

	hipMemcpy2D(
				*pLpvBits,						// Destination 
				src_pitch,			
				cuda_lpvbits,					// Source
				dest_pitch,		
				WIDTH * 4 * sizeof(BYTE),		
				HEIGHT, 
				hipMemcpyDeviceToHost);


	hipFree(cuda_cam);
	hipFree(cuda_lpvbits);
	
}
